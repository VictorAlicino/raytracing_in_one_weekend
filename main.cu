#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <time.h>

#include "vec3.cuh"
#include "ray.cuh"
#include "hitable_list.cuh"
#include "hitable.cuh"
#include "sphere.cuh"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f*(unit_direction.y() + 1.0f);
        return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        col += color(r, world);
    }
    fb[pixel_index] = col/float(ns);
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *d_world    = new hitable_list(d_list,2);
        *d_camera   = new camera();
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char **argv) {
    int nx = atoi(argv[1]); // Image Width
    int ny = atoi(argv[2]); // Image Height
    int tx = 8;
    int ty = 8;

    std::clog << "Rendering a " << nx << "x" << ny << " image ";
    std::clog << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    std::clog << "Allocating Frame Buffer\n";
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // make our world of hitables
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    create_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    std::clog << "Starting GPU Kernel\n";
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny,
                                vec3(-2.0, -1.0, -1.0),
                                vec3(4.0, 0.0, 0.0),
                                vec3(0.0, 2.0, 0.0),
                                vec3(0.0, 0.0, 0.0),
                                d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::clog << "GPU's done computing image";
    std::cerr << " -> took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::clog << "Sending Image to the file\n";
    start = clock();
    // Open the output file stream
    std::string output = std::string(argv[3]) + ".ppm";
    std::ofstream outfile(output);
    // Write the PPM header
    outfile << "P3\n" << nx << " " << ny << "\n255\n";

    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            // Write pixel color to the file stream
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }

    // Close the output file stream
    outfile.close();
    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::clog << "Image sent to SDOUT";
    std::cerr << " -> took " << timer_seconds << " seconds.\n";

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}